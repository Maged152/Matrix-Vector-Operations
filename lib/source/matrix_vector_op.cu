#include "types.hpp"

qlm::DeviceMemory::DeviceMemory(size_t size) : size(size)
{
    hipMalloc(&data, size * sizeof(float));
}
qlm::DeviceMemory::~DeviceMemory()
{
    if (data != nullptr) {
        hipFree(data);
        data = nullptr;
    }
}

void qlm::DeviceMemory::ToCPU(float *hostData) const
{
    hipMemcpy(hostData, data, size * sizeof(float), hipMemcpyDeviceToHost);
}

void qlm::DeviceMemory::FromCPU(const float *hostData)
{
    hipMemcpy(data, hostData, size * sizeof(float), hipMemcpyHostToDevice);
}