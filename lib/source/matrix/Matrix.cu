#include "matrix_vector_op.hpp"
#include <hiprand/hiprand_kernel.h>


namespace qlm
{
    // Default constructor
	Matrix::Matrix()
	{}

	// Parameterized constructor
	Matrix::Matrix(int rows, int columns) : rows(rows), columns(columns) 
	{
		hipMalloc(&data, rows * columns * sizeof(float));
	}

	// Copy constructor
	Matrix::Matrix(const Matrix& other) : columns(other.columns), rows(other.rows) 
	{
		hipMalloc(&data, rows * columns * sizeof(float));
        hipMemcpy(data, other.data, rows * columns * sizeof(float), hipMemcpyDeviceToDevice);
	}

	// Destructor
	Matrix::~Matrix() 
	{
		rows = columns = stride = 0;
		if (data != nullptr)
			hipFree(data);
	}

	// Setter for individual element (host to device)
	void Matrix::Set(int row, int col, float value) 
	{
		if (row >= 0 && row < rows && col >= 0 && col < columns)
		{
            data[row * columns + col] = value;
		}
	}

	// Getter for individual element
	float Matrix::Get(int row, int col) const 
	{
        float value = -1.0f; 
		if (row >= 0 && row < rows && col >= 0 && col < columns)
		{
			return data[row * columns + col];
		}
		return value;
	}

	// Getter for columns
	int Matrix::Columns() const
	{
		return columns;
	}

	// Getter for rows
	int Matrix::Rows() const
	{
		return rows;
	}

    int Matrix::Stride() const
	{
		return stride;
	}

    void Matrix::Alloc(const int rows, const int columns)
    {
        if (data != nullptr)
            hipFree(data);
        this->rows = rows;
        this->columns = columns;
        hipMalloc(&data, rows * columns * sizeof(float));
    }
    
    void Matrix::FromCPU(const float* src, const int num_rows, const int num_columns)
    {
		if (rows != num_rows || columns != num_columns || data == nullptr)
        	Alloc(num_rows, num_columns);

		hipMemcpy(data, src, rows * columns * sizeof(float), hipMemcpyHostToDevice);
    }
    
	void Matrix::ToCPU(float* dst, const int rows, const int columns) const
    {
        if (dst != nullptr && data != nullptr)
        {
			hipMemcpy(dst, data, rows * columns * sizeof(float), hipMemcpyDeviceToHost);
        }
    }
}