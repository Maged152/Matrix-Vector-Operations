#include "types.hpp"

qlm::DeviceBuffer::DeviceBuffer(size_t size) : size(size)
{
    hipMalloc(&data, size * sizeof(float));
}
qlm::DeviceBuffer::~DeviceBuffer()
{
    if (data != nullptr) {
        hipFree(data);
        data = nullptr;
    }
}

void qlm::DeviceBuffer::ToCPU(float *hostData) const
{
    hipMemcpy(hostData, data, size * sizeof(float), hipMemcpyDeviceToHost);
}

void qlm::DeviceBuffer::FromCPU(const float *hostData)
{
    hipMemcpy(data, hostData, size * sizeof(float), hipMemcpyHostToDevice);
}