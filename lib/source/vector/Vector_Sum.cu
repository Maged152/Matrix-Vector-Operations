#include "hip/hip_runtime.h"
#include "matrix_vector_op.hpp"

#define BLOCK_SIZE 256

namespace qlm
{
    __device__ void WarpReduce(volatile float* partial_sum, const int tid) 
    {
        partial_sum[tid] += partial_sum[tid + 32];
        partial_sum[tid] += partial_sum[tid + 16];
        partial_sum[tid] += partial_sum[tid + 8];
        partial_sum[tid] += partial_sum[tid + 4];
        partial_sum[tid] += partial_sum[tid + 2];
        partial_sum[tid] += partial_sum[tid + 1];
    }

    __global__ void VectorSum_Cuda(const float* in, const int length, float* result)
    {
        const int tid = threadIdx.x;
        const int gid = blockIdx.x * blockDim.x * 2 + tid;

        __shared__ float partial_sum[BLOCK_SIZE];

        // Load elements & do first add of reduction
        const float second_element = (gid + blockDim.x < length) ? in[gid + blockDim.x] : 0.0f;
        partial_sum[tid] = in[gid] + second_element;
        __syncthreads();

        for (int s = blockDim.x / 2; s > 32; s >>= 1) 
        { 
            if (tid < s)
            {
                partial_sum[tid] += partial_sum[tid + s];
            }
            __syncthreads();
        }

        // last warp
        if (tid < 32) 
            WarpReduce(partial_sum, tid);

        if (tid == 0) 
        {
            result[blockIdx.x] = partial_sum[0];
        }
    }

    __global__ void VectorSumBlock_Cuda(const float* in, const int length, float* result)
    {
        const int tid = threadIdx.x;

        __shared__ float partial_sum[BLOCK_SIZE];
        partial_sum[tid] = (tid < length) ? in[tid] : 0.0f;

	    __syncthreads();

        for (int s = blockDim.x / 2; s > 32; s >>= 1) 
        {
            if (tid < s)
            {
                partial_sum[tid] += partial_sum[tid + s];
            }
            __syncthreads();
        }

        // last warp
        if (tid < 32) 
            WarpReduce(partial_sum, tid);


        if (tid == 0) 
        {
            *result += partial_sum[0];
        }
    }


    void qlm::Sum(const Vector &src, DeviceFloat& result)
	{
        const int length = src.Length();
        // Launch kernel
        const int block_size = BLOCK_SIZE;
        const int num_blocks = (length + (block_size * 2) - 1) / (block_size * 2);

        // allocate device memory for the result
        float* sum_result;
        hipMalloc(&sum_result, num_blocks * sizeof(float));
        
        // First reduction: input -> partial sums
        VectorSum_Cuda<<<num_blocks, block_size>>>(src.data, length, sum_result);
        hipDeviceSynchronize();

        // Second reduction: partial sums -> final sum
        for (int i = 0; i < num_blocks; i += block_size) 
        {
            const int cur_length = std::min(block_size, num_blocks - i);
            VectorSumBlock_Cuda<<<1, block_size>>>(&sum_result[i], cur_length, result.mem.data);
            hipDeviceSynchronize();
        }
        
        hipFree(sum_result); // Free the device memory
	}
}