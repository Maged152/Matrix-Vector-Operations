#include "hip/hip_runtime.h"
#include "vector.hpp"

namespace qlm
{
    __global__ void VectorAdd_Cuda(const float* in0, const float* in1, float* out, const int length)
    {
        const int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < length)
        {
            out[tid] = in0[tid] + in1[tid];
        }
    }

    void qlm::Vector::Add(const Vector &src, Vector &dst) const
	{
        // Launch kernel
        const int block_size = 256;
        const int num_blocks = (length + block_size - 1) / block_size;
        VectorAdd_Cuda<<<num_blocks, block_size>>>(data, src.data, dst.data, length);
        hipDeviceSynchronize(); // Ensure the kernel execution is complete
	}
}